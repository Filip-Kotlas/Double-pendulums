#include "hip/hip_runtime.h"
#include "System.hpp"
#include <stdexcept>
#include <cmath>
#include <map>

const std::vector<double>& System::get_state_history(double time) {
    if (state_history.empty()) {
        throw std::runtime_error("State history is empty");
    }

    auto lower_bound = this->state_history.lower_bound(time);

    if (lower_bound == state_history.begin())
        return lower_bound->second;
    else if (lower_bound == state_history.end())
        return std::prev(lower_bound)->second;
    else {
        auto previous = std::prev(lower_bound);
        if (std::abs(previous->first - time) < std::abs(lower_bound->first - time))
            return previous->second;
        else
            return lower_bound->second;
    }
}