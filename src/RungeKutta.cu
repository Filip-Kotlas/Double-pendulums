#include "hip/hip_runtime.h"
#include "RungeKutta.hpp"

void RungeKutta::set_up(System *system, double time_step, double integration_step)
{
    int dof = system->get_degrees_of_freedom();
    k1.resize(dof);
    k2.resize(dof);
    k3.resize(dof);
    k4.resize(dof);
    aux.resize(dof);

    this->time_step = time_step;
    this->integration_step = integration_step;
    this->current_system = system;
}

void RungeKutta::solve(double time_max, std::string output_folder)
{
    int steps_count = std::round((time_max - this->current_system->get_time())/this->time_step);
    this->current_system->record_state();
    auto clock_computation_start = std::chrono::high_resolution_clock::now();
    this->current_system->write_state_to_file(this->current_system->get_time(), this->time_step, output_folder);

    for(int k = 1; k <= steps_count; k++){
        auto clock_step_start = std::chrono::high_resolution_clock::now();
        this->integrate_step(time_max);
        this->current_system->copy_data_from_device_to_host();
        this->current_system->record_state();
        this->current_system->write_state_to_file(this->current_system->get_time(), time_step, output_folder);
        auto clock_step_end = std::chrono::high_resolution_clock::now();

        double time_per_step = (std::chrono::duration<double>(clock_step_end - clock_step_start).count());
        double remaining_time = time_per_step * (steps_count - k);
        int hours_remaining = static_cast<int>(remaining_time) / 3600;
        int minutes_remaining = (static_cast<int>(remaining_time) % 3600) / 60;
        int seconds_remaining = remaining_time - (hours_remaining * 3600 + minutes_remaining * 60);

        double elapsed_time = (std::chrono::duration<double>(clock_step_end - clock_computation_start).count());
        int hours_elapsed = static_cast<int>(elapsed_time) / 3600;
        int minutes_elapsed = (static_cast<int>(elapsed_time) % 3600) / 60;
        int seconds_elapsed = elapsed_time - (hours_elapsed * 3600 + minutes_elapsed * 60);

        std::cout << "Steps completed: " << k << " / " << steps_count << " => " << std::fixed
                  << std::setprecision(2) << (double) k / (double) steps_count * 100.0 << "% ";
        std::cout << "     Time elapsed: " << hours_elapsed << "h " << minutes_elapsed << "m "
                  << seconds_elapsed << "s";
        std::cout << "     Time remaining: " << hours_remaining << "h " << minutes_remaining << "m "
                  << seconds_remaining << "s" << std::endl;
    }
}

void RungeKutta::integrate_step(double time_max)
{
    double start_time = current_system->get_time();
    int cuda_block_size = 256;
    int cuda_grid_size = 88;
    while (this->integration_step <= start_time + this->time_step - current_system->get_time()) {
        double tau = std::min(this->integration_step, start_time + this->time_step - current_system->get_time());

        // Computing k1
        current_system->get_right_hand_side(current_system->get_time(),
                                            current_system->get_cuda_state(),
                                            k1);
        
        // Computing k2
        cuda_aux_computation<<<cuda_grid_size, cuda_block_size>>>(current_system->get_cuda_state().data(),
                                                                  k1.data(),
                                                                  aux.data(),
                                                                  tau,
                                                                  current_system->get_degrees_of_freedom(),
                                                                  0.5);
        hipDeviceSynchronize();
        current_system->get_right_hand_side(current_system->get_time() + 1.0/2*tau,
                                            aux,
                                            k2);
        
        // Computing k3
        cuda_aux_computation<<<cuda_grid_size, cuda_block_size>>>(current_system->get_cuda_state().data(),
                                                                  k2.data(),
                                                                  aux.data(),
                                                                  tau,
                                                                  current_system->get_degrees_of_freedom(),
                                                                  0.5);
        hipDeviceSynchronize();
        current_system->get_right_hand_side(current_system->get_time() + 1.0/2*tau,
                                            aux,
                                            k3);

        // Computing k4
        cuda_aux_computation<<<cuda_grid_size, cuda_block_size>>>(current_system->get_cuda_state().data(),
                                                                  k3.data(),
                                                                  aux.data(),
                                                                  tau,
                                                                  current_system->get_degrees_of_freedom(),
                                                                  1.0);
        hipDeviceSynchronize();
        current_system->get_right_hand_side(current_system->get_time() + tau,
                                            aux,
                                            k4);

        // Final update
        cuda_final_update<<<cuda_grid_size, cuda_block_size>>>(current_system->get_cuda_state().data(),
                                                                k1.data(),
                                                                k2.data(),
                                                                k3.data(),
                                                                k4.data(),
                                                                tau,
                                                                current_system->get_degrees_of_freedom());
        hipDeviceSynchronize();
        current_system->increase_time(tau);
    }
}

__global__ void cuda_aux_computation(const double* state, const double* k, double* aux, double tau, int dof, double factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < dof; i += stride) {
        if (i < dof) {
            aux[i] = state[i] + factor * tau * k[i];
        }
    }
}

__global__ void cuda_final_update(double* state, const double* k1, const double* k2, const double* k3, const double* k4, double tau, int dof) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < dof; i += stride) {
        if (i < dof) {
            state[i] += 1.0/6 * tau * (k1[i] + 2*k2[i] + 2*k3[i] + k4[i]);
        }
    }
}