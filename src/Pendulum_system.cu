#include "hip/hip_runtime.h"
#include "Pendulum_system.hpp"

void PendulumSystem::get_right_hand_side(const double time, const CudaArray &state, CudaArray &right_hand_side)
{
    dim3 grid_size = 256;
    dim3 block_size = 256;
    cuda_right_hand_side<<<block_size, grid_size>>>(state.data(), right_hand_side.data(), this->size_x * this->size_y);
    hipDeviceSynchronize();
}

void PendulumSystem::set_initial_conditions(const double time)
{
    for(int i = 0; i < this->size_x; i++){
        for(int j = 0; j < this->size_y; j++){
            set_der_phi_1(i, j, 0);
            set_der_phi_2(i, j, 0);
            set_phi_1(i, j, bounds[0] + (i + 1)*(bounds[1] - bounds[0])/(size_x + 1));
            set_phi_2(i, j, bounds[2] + (j + 1)*(bounds[3] - bounds[2])/(size_y + 1));
        }
    }
}

void PendulumSystem::write_state_to_file(double save_time, double time_step, std::string folder_name)
{
    std::filesystem::path results_path("results");
    std::filesystem::path dir_path = results_path / folder_name;
    std::filesystem::create_directories(dir_path);
    std::stringstream file_name;
    file_name << "State_" << std::setw(5) << std::setfill('0') << std::round(save_time/time_step) << ".txt";
    std::filesystem::path file_path = dir_path / file_name.str();
    
    std::fstream file;
    file.open( file_path, std::fstream::out | std::fstream::trunc );
    if(!file)
    {
        throw std::ios_base::failure("Unable to open the file: " + file_path.string());
    }

    file << std::scientific << std::setprecision(15);
    file << save_time << std::endl << std::endl;
    file << this->size_x << std::endl << this->size_y << std::endl << std::endl;
    for(int j = 0; j < size_y; j++)
    {
        for( int i = 0; i < size_x; i++ )
        {
            file << i << " "
                 << j << " "
                 << get_phi_1(i, j, save_time) << " "
                 << get_phi_2(i, j, save_time) << " "
                 << get_der_phi_1(i, j, save_time) << " "
                 << get_der_phi_2(i, j, save_time);
            file << std::endl;
        }
        file << std::endl;
    }
    file.close();
}

void PendulumSystem::record_state()
{
    this->state_history[this->time] = host_state;
}

__global__ void cuda_right_hand_side(const double* state,
                                     double* right_hand_side,
                                     int size,
                                     double mass_1,
                                     double mass_2,
                                     double length_1,
                                     double length_2)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for ( int i = tid; i < size; i += stride)
    {
        float g = 9.81;
        double a, b, c, d, e, f;
    
        a = -(mass_1 + mass_2)*g*length_1*sin(state[i*4])
            - mass_2*length_1*length_2*pow(state[i*4 + 3], 2)*sin(state[i*4] - state[i*4 + 1]);
        b = (mass_1 + mass_2)*length_1*length_1;
        c = mass_2*length_1*length_2*cos(state[i*4] - state[i*4 + 1]);
        d = -mass_2*g*length_2*sin(state[i*4 + 1])
            + mass_2*length_1*length_2*pow(state[i*4 + 2], 2)*sin(state[i*4] - state[i*4 + 1]);
        e = mass_2*length_2*length_2;
        f = mass_2*length_1*length_2*cos(state[i*4] - state[i*4 + 1]);
    
        right_hand_side[i*4] = state[i*4 + 2];
        right_hand_side[i*4 + 1] = state[i*4 + 3];
        right_hand_side[i*4 + 2] = a/b - c/b*(d-a*f/b)/(e-c*f/b);
        right_hand_side[i*4 + 3] = (d-a*f/b)/(e-c*f/b);
    }
}